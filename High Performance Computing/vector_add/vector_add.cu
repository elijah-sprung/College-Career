
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

__host__ void vector_add(float* vector_a, float* vector_b, float* vector_c, int length);

__global__ void vecAddKernel(float* A, float* B, float* C, int length);

void load_vector(float** vector, int* length, char *file_name);
void output_vector(float** vector, int length, char *file_name);

int main(int argc, char* argv[])
{
	if (argc != 4)
	{
		fprintf(stderr, "Invalid number of arguments. \n");
		exit(1);
	}

	float * vector_a = NULL, * vector_b = NULL, * vector_c = NULL;
	int length;

	load_vector(&vector_a, &length, argv[1]);
	load_vector(&vector_b, &length, argv[2]);

	vector_c = (float*)malloc(length * sizeof(float));

	vector_add(vector_a, vector_b, vector_c, length);

	output_vector(&vector_c, length, argv[3]);

	return 0;
}

__host__ void vector_add(float* vector_a, float* vector_b, float* vector_c, int length)
{
	float *A_d, *B_d, * C_d;
	int size = length * sizeof(float);

	hipMalloc((void**)&A_d, size);
	hipMalloc((void**)&B_d, size);
	hipMalloc((void**)&C_d, size);

	hipMemcpy(A_d, vector_a, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, vector_b, size, hipMemcpyHostToDevice);

	vecAddKernel<<<ceil(length / 256.0), 256 >>>(A_d, B_d, C_d, length);

	hipMemcpy(vector_c, C_d, size, hipMemcpyDeviceToHost);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}

__global__ void vecAddKernel(float* A, float* B, float* C, int length)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < length)
	{
		C[i] = A[i] + B[i];
	}
}

void load_vector(float** vector, int* length, char *file_name)
{
	FILE* input_file;
	float holder;

	if ((input_file = fopen(file_name, "r")) == NULL)
	{
		printf("Error: File could not be opened. \n");
		exit(2);
	}

	fscanf(input_file, "%f", &holder);
	// printf("%f", holder);

	*length = holder;

	*vector = (float*)malloc(*length * sizeof(float));

	for (int i = 0; i < *length; i++)
	{
		fscanf(input_file, "%f", &holder);
		(*vector)[i] = holder;
	}

	fclose(input_file);
}

void output_vector(float** vector, int length, char *file_name)
{
	FILE* output_file;

	if ((output_file = fopen(file_name, "w")) == NULL)
	{
		printf("Error: File could not be opened. \n");
		exit(2);
	}

	for (int i = 0; i < length; i++)
	{
		fprintf(output_file, "%f ", (*vector)[i]);
	}

	fclose(output_file);
}
